#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void func(int *dataArr,int *x,int *ptr,int *y,int *index) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int val = 0;
	for(int j = ptr[i]; j < ptr[i+1]; j++) {
		val = val + dataArr[j]*x[index[j]];
	}
	y[i] = val;
} 

int main() {
	int m,n,i,j;
	int *d_a,*d_x,*d_y,*d_ptr,*d_index;

	printf("Enter value of m: ");
	scanf("%d",&m);
	printf("Enter value of n: ");
	scanf("%d",&n);

	int size = sizeof(int);
	int a[m][n];
	int x[n];
	int y[n];
	int ptr[100];
	int ptrid = 0;
	int index[100];
	int indexid = 0;
	int dataArr[100];
	int temp,f;

	printf("Enter input matrix: ");
	for(i = 0; i<m; i++) {
		f = 0;
		for(j = 0; j<n; j++) {
			scanf("%d",&a[i][j]);
			if(a[i][j] != 0) {
				if(f == 0)
					temp = indexid;
				f = 1;
				index[indexid] = j;
				dataArr[indexid] = a[i][j];
				indexid++;
			}
		}		
		if(f == 1) {
			ptr[ptrid] = temp;
			ptrid++;
		}
	}
	ptr[ptrid] = indexid;
	ptrid++;

	printf("Enter input vector: ");
	for(i = 0; i<n; i++) {
		scanf("%d",&x[i]);		
	}

	hipMalloc((void **)&d_a,size*indexid);
	hipMalloc((void **)&d_x,size*n);
	hipMalloc((void **)&d_y,size*n);
	hipMalloc((void **)&d_ptr,size*ptrid);
	hipMalloc((void **)&d_index,size*indexid);

	hipMemcpy(d_a,dataArr,size*indexid,hipMemcpyHostToDevice);
	hipMemcpy(d_x,x,size*n,hipMemcpyHostToDevice);
	hipMemcpy(d_ptr,ptr,size*ptrid,hipMemcpyHostToDevice);
	hipMemcpy(d_index,index,size*indexid,hipMemcpyHostToDevice);

	func<<<1,n>>>(d_a,d_x,d_ptr,d_y,d_index);

	hipMemcpy(y,d_y,size*n,hipMemcpyDeviceToHost);

	printf("Result vector is: \n");
	for(j = 0; j < n; j++) {
		printf("%d\t",y[j]);
	}
	printf("\n");

	hipFree(d_a);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_ptr);
	hipFree(d_index);
	return 0;
}