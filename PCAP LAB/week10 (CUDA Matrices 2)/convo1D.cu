#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void func(float *N,float *M,float *P,int mwidth, int width) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float pval = 0.0;
	int N_start_point = i - (mwidth/2);
	for(int j = 0; j < mwidth; j++) {
		if((N_start_point + j) >= 0 && (N_start_point + j) < width) {
			pval = pval + N[N_start_point + j] * M[j];
			//printf("Multiplying: %f and %f\n",N[N_start_point + j],M[j]);
		}
	}
	P[i] = pval;
} 

int main() {
	int i,j;
	float *d_n,*d_m,*d_p;

	int width,mwidth;

	printf("Enter size of input: ");
	scanf("%d",&width);
	printf("Enter size of mask (has to be odd): ");
	scanf("%d",&mwidth);

	int size = sizeof(float);
	float n[width];
	float m[mwidth];

	printf("Enter input vector: ");
	for(i = 0; i<width; i++) {
		scanf("%f",&n[i]);
	}

	printf("Enter mask: ");
	for(i = 0; i<mwidth; i++) {
		scanf("%f",&m[i]);
	}

	float p[width];

	hipMalloc((void **)&d_n,size*width);
	hipMalloc((void **)&d_m,size*mwidth);
	hipMalloc((void **)&d_p,size*width);

	hipMemcpy(d_n,n,size*width,hipMemcpyHostToDevice);
	hipMemcpy(d_m,m,size*mwidth,hipMemcpyHostToDevice);

	dim3 dimGrid(((width - 1) / mwidth) + 1,1,1);
	dim3 dimBlock(mwidth,1,1);
	func<<<dimGrid,dimBlock>>>(d_n,d_m,d_p,mwidth,width);

	hipMemcpy(p,d_p,size*width,hipMemcpyDeviceToHost);

	printf("Result vector is: \n");
	for(j = 0; j <width; j++) {
		printf("%4f\n",p[j]);
	}
	printf("\n");

	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_p);
	return 0;
}