#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define BW 2
#define TW 2
#define W 4

__device__ int getTid() {
	int blockSkip = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y);
	int rowSkip = (threadIdx.y * gridDim.x * blockDim.x);
	int rowDisp = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tid = blockSkip + rowSkip + rowDisp;
	return tid;
}

__global__ void func(int *a,int *b,int *t) {
	int tid = getTid();
	t[tid] = a[tid] + b[tid];
} 

int main() {
	int i,j;
	int *d_a,*d_b,*d_t;

	//m = n = 4
	int m,n;
	m = n = W;

	int size = sizeof(int)*n*m;
	int a[m][n];
	int b[m][n];
	int t[m][n];

	printf("Enter input matrix 1: ");
	for(i = 0; i<m; i++) {
		for(j = 0; j<n; j++) {
			scanf("%d",&a[i][j]);
		}		
	}

	printf("Enter input matrix 2: ");
	for(i = 0; i<m; i++) {
		for(j = 0; j<n; j++) {
			scanf("%d",&b[i][j]);
		}		
	}

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	int numblocks = W / BW;
	dim3 dimGrid(numblocks,numblocks,1);
	dim3 dimBlock(BW,BW,1);
	func<<<dimGrid,dimBlock>>>(d_a,d_b,d_t);

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);

	printf("Result matrix is: \n");
	for(i = 0; i < m; i++) {
		for(j = 0; j < n; j++) {
			printf("%d\t",t[i][j]);
		}
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_t);
	return 0;
}