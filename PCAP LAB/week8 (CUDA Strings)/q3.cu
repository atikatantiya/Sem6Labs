#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void func(char *a,char *B,unsigned int *d_count,int len,int wordlen) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int start = 0;
	int end = 0;
	for(int j=0; j<len; ++j) {
	  if(a[j] == ' '){
	    id--;
	    if(id == 0) 
	      start = j+1;
	    if(id < 0){
	      end = j-1;
	      break;
	    }
	  }
	}
	if((end - start + 1) == wordlen) {
		int f = 1;
		int j = 0;
		while(start < end) {
			if(a[start] != B[j]) {
				f = 0;
				break;
			}
			j++;
			start++;		
		}
		if(f == 1)
			atomicAdd(d_count,1);
	}	
}

int main() {
	char a[500],b[100];

	char *d_A,*d_B;
	unsigned int count = 0, *d_count, result;

	printf("Enter string: ");
	gets(a);
	int len = strlen(a);
  	a[len++] = ' ';

	printf("Enter number of words: ");
	int n;
	scanf("%d",&n);

	gets(b);
	printf("Enter word: ");
	gets(b);
	int wordlen = strlen(b);

	hipMalloc((void **)&d_A,strlen(a)*sizeof(char));
	hipMalloc((void **)&d_B,strlen(b)*sizeof(char));
	hipMalloc((void **)&d_count,sizeof(unsigned int));

	hipMemcpy(d_A,a,strlen(a)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_B,b,strlen(b)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(unsigned int),hipMemcpyHostToDevice);

	func<<<1,n>>>(d_A,d_B,d_count,len,wordlen);	

	hipMemcpy(&result,d_count,sizeof(unsigned int),hipMemcpyDeviceToHost);

	printf("Total occurences of %s: %u\n",b,result);	

	hipFree(d_A);
	hipFree(d_count);
}