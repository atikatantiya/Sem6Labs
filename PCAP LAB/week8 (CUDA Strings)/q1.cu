#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<>
#include<string.h>

__global__ void conc(char *a,int len,char *b) {
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  for(int i=0; i<len; i++)
    b[id*len+i] = a[i];
}

int main() {
  int n,len;
  char a[100],b[500];

  printf("Enter string:");
  gets(a);

  printf("Enter n: ");
  scanf("%d",&n);

  len = strlen(a);

  char *d_a,*d_b;
  memset(b,0,100);

  hipMalloc((void**)&d_a,len);
  hipMalloc((void**)&d_b,len*n);

  hipMemcpy(d_a,&a,len,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,len*n,hipMemcpyHostToDevice);

  conc<<<1,n>>>(d_a,len,d_b);

  hipMemcpy(&b,d_b,len*n,hipMemcpyDeviceToHost);

  printf("%s\n",b);

  hipFree(d_a);
  hipFree(d_b);
}
