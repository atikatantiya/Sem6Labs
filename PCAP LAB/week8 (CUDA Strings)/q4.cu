
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void reverse (char* str, char* res, int N, int len) {
    int numWords = threadIdx.x+1;
    int start = 0, end = 0;
    for(int i=0; i<len; ++i){
        if(str[i] == ' '){
            numWords--;
            if(numWords == 1) 
                start = i;
            else if(numWords <= 0){
                end = i;
                break;
            }
        }
    }
    for(int i=start; i<end;++i) 
        res[len-i-1] = str[i];
}

int main(){
    char string[100], res[100];
    int N, len;
    printf("Enter the string: ");
    fgets(string, 100, stdin);
    printf("Enter the number of words: ");
    scanf("%d", &N);
    
    char *d_str, *d_res;
    len = strlen(string);
    string[len-1] = ' ';

    hipMalloc((void**)&d_str, sizeof(char)*(len+1));
    hipMalloc((void**)&d_res, sizeof(char)*(len+1));

    hipMemcpy(d_str, string, len, hipMemcpyHostToDevice);

    reverse<<<1, N>>>(d_str, d_res, N, len);
    
    hipMemcpy(res, d_res, len, hipMemcpyDeviceToHost);

    res[len] = 0;

    printf("The result is: %s\n", res);

    for(int i=0; i<len; ++i) 
        printf("%c", res[i]);
    return 0;
}
