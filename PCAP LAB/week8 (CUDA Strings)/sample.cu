#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void func(char *A,unsigned int *d_count) {
	int id = threadIdx.x;
	if(A[id] == 'a')
		atomicAdd(d_count,1);
}

int main() {
	char A[N];

	char *d_A;
	unsigned int count = 0, *d_count, result;

	printf("Enter string: ");
	gets(A);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	hipMalloc((void **)&d_A,strlen(A)*sizeof(char));
	hipMalloc((void **)&d_count,sizeof(unsigned int));

	hipMemcpy(d_A,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(unsigned int),hipMemcpyHostToDevice);

	func<<<1,strlen(A)>>>(d_A,d_count);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsed;
	hipEventElapsedTime(&elapsed,start,stop);

	hipMemcpy(&result,d_count,sizeof(unsigned int),hipMemcpyDeviceToHost);

	printf("Total occurences of a: %u\n",result);
	printf("Time taken: %f\n",elapsed);

	hipFree(d_A);
	hipFree(d_count);
}