#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<>
#include<string.h>

__global__ void conc(char *a,int len) {
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = 0;
  int end = 0;
  for(int j=0; j<len; ++j) {
    if(a[j] == ' '){
      id--;
      if(id == 0) 
        start = j+1;
      if(id < 0){
        end = j-1;
        break;
      }
    }
  }
  while(start < end){
    id = a[start];
    a[start++] = a[end];
    a[end--] = id;
  }
}

int main() {
  int n,len;
  char a[100],b[100];

  printf("Enter str:");
  gets(a);

  printf("Enter n: ");
  scanf("%d",&n);

  len = strlen(a);
  a[len++] = ' ';

  char *d_a;
  memset(b,0,100);

  hipMalloc((void**)&d_a,len);

  hipMemcpy(d_a,&a,len,hipMemcpyHostToDevice);

  conc<<<1,n>>>(d_a,len);

  hipMemcpy(&a,d_a,len,hipMemcpyDeviceToHost);

  printf("%s\n",a);

  hipFree(d_a);
}
