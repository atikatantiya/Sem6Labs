#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void func(int *a,int *b,int *t,int rows,int cols) {
	int id = threadIdx.y;
	for(int i = 0; i < rows; i++) {
		t[id] =  a[id] + b[id];
		id = id + cols;
	}
} 

int main() {
	int m,n,i,j;
	int *d_a,*d_b,*d_t;

	printf("Enter value of m: ");
	scanf("%d",&m);
	printf("Enter value of n: ");
	scanf("%d",&n);

	int size = sizeof(int)*n*m;
	int a[m][n];
	int b[m][n];
	int t[m][n];

	printf("Enter input matrix 1: ");
	for(i = 0; i<m; i++) {
		for(j = 0; j<n; j++) {
			scanf("%d",&a[i][j]);
		}		
	}

	printf("Enter input matrix 2: ");
	for(i = 0; i<m; i++) {
		for(j = 0; j<n; j++) {
			scanf("%d",&b[i][j]);
		}		
	}

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	dim3 dimGrid(1,1,1);
	dim3 dimBlock(1,n,1);
	func<<<dimGrid,dimBlock>>>(d_a,d_b,d_t,m,n);

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result matrix is: \n");
	for(i = 0; i < m; i++) {
		for(j = 0; j < n; j++) {
			printf("%d\t",t[i][j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_t);
	return 0;
}