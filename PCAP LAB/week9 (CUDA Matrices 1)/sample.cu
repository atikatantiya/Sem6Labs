#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void transpose(int *a,int *t) {
	int n = threadIdx.x;
	int m = blockIdx.x;
	int size = blockDim.x;
	int size1 = gridDim.x;
	t[n*size1 + m] = a[m*size + n];
}

int main() {
	int *a,*t,m,n,i,j;
	int *d_a,*d_t;

	printf("Enter value of m: ");
	scanf("%d",&m);

	printf("Enter value of n: ");
	scanf("%d",&n);

	int size = sizeof(int)*m*n;
	a = (int *)malloc(sizeof(int)*m*n);
	t = (int *)malloc(sizeof(int)*m*n);

	printf("Enter input matrix: ");
	for(i = 0; i<m*n; i++) {
		scanf("%d",&a[i]);
	}

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

	transpose<<<m,n>>>(d_a,d_t);

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result matrix is: ");
	for(i = 0; i < m; i++) {
		for(j = 0; j < n; j++) {
			printf("%d\t",t[i*m + j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}