#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void mult(int *a,int *b,int *c,int ha,int wa,int wb) {
  int id = threadIdx.x;
  int sum = 0;
  for(int i=0; i<ha; i++) {
    sum = 0;
    for(int j=0; j<wa; j++)
      sum = sum + a[i*wa+j]*b[j*wb+id];
    c[i*wb+id] = sum;
  }
}

int main() {
  int ha,wa,wb; 
  printf("Enter ha,wa,wb: ");
  scanf("%d %d %d",&ha,&wa,&wb);

  int a[ha][wa],b[wa][wb]; 
  int c[ha][wb];

  printf("Enter A:\n");
  for(int i=0; i<ha; i++){
    for(int j = 0; j < wa; j++) {
        scanf("%d",&a[i][j]);
    }
  }

  printf("Enter B:\n");
  for(int i=0; i<wa; i++){
    for(int j = 0; j < wb; j++) {
        scanf("%d",&b[i][j]);
    }
  }

  int *d_a,*d_b,*d_c;
  int size = sizeof(int); 

  hipMalloc((void**)&d_a,size*ha*wa);
  hipMalloc((void**)&d_b,size*wa*wb);
  hipMalloc((void**)&d_c,size*ha*wb);

  hipMemcpy(d_a,&a,size*ha*wa,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size*wa*wb,hipMemcpyHostToDevice);

  mult<<<1,wb>>>(d_a,d_b,d_c,ha,wa,wb);

  hipMemcpy(&c,d_c,size*ha*wb,hipMemcpyDeviceToHost);

  printf("C:\n");
  for(int i=0; i<ha; i++) {
    for(int j=0; j<wb; j++)
      printf("%d ",c[i][j]);
    printf("\n");
  }
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
