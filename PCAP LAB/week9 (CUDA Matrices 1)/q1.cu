#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void func(int *a,int *t) {
	int id = threadIdx.y*blockDim.x + threadIdx.x;
	int power = threadIdx.y + 1;
	t[id] = 1;
	for(int i = 1; i <= power; i++)
		t[id] = t[id] * a[id];
}

int main() {
	int n,i,j;
	int *d_a,*d_t;

	printf("Enter value of n: ");
	scanf("%d",&n);

	int size = sizeof(int)*n*n;
	int a[n][n];
	int t[n][n];

	printf("Enter input matrix: ");
	for(i = 0; i<n; i++) {
		for(j = 0; j<n; j++) {
			scanf("%d",&a[i][j]);
		}		
	}

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

	dim3 dimGrid(1,1,1);
	dim3 dimBlock(n,n,1);
	func<<<dimGrid,dimBlock>>>(d_a,d_t);

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result matrix is: \n");
	for(i = 0; i < n; i++) {
		for(j = 0; j < n; j++) {
			printf("%d\t",t[i][j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}