#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void func(int *a,int *b,int *t,int cols) {
	int id = threadIdx.x*cols;
	for(int i = 1; i <= cols; i++) {
		t[id] =  a[id] + b[id];
		id++;
	}
} 

int main() {
	int m,n,i,j;
	int *d_a,*d_b,*d_t;

	printf("Enter value of m: ");
	scanf("%d",&m);
	printf("Enter value of n: ");
	scanf("%d",&n);

	int size = sizeof(int)*n*m;
	int a[m][n];
	int b[m][n];
	int t[m][n];

	printf("Enter input matrix 1: ");
	for(i = 0; i<m; i++) {
		for(j = 0; j<n; j++) {
			scanf("%d",&a[i][j]);
		}		
	}

	printf("Enter input matrix 2: ");
	for(i = 0; i<m; i++) {
		for(j = 0; j<n; j++) {
			scanf("%d",&b[i][j]);
		}		
	}

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	func<<<1,m>>>(d_a,d_b,d_t,n);

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result matrix is: \n");
	for(i = 0; i < m; i++) {
		for(j = 0; j < n; j++) {
			printf("%d\t",t[i][j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_t);
	return 0;
}