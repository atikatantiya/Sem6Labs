#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void mult(int *a,int *b,int *c,int wa,int wb) {
  int col = threadIdx.x;
  int row = threadIdx.y;
  int sum = 0;
  for(int i=0; i<wa; i++)
    sum = sum + a[row*wa+i]*b[i*wb+col];
  c[row*wb+col] = sum;
}

int main() {
  int ha,wa,wb; 
  printf("Enter ha,wa,wb: ");
  scanf("%d %d %d",&ha,&wa,&wb);

  int a[ha][wa],b[wa][wb]; 
  int c[ha][wb];

  printf("Enter A:\n");
  for(int i=0; i<ha; i++){
    for(int j = 0; j < wa; j++) {
        scanf("%d",&a[i][j]);
    }
  }

  printf("Enter B:\n");
  for(int i=0; i<wa; i++){
    for(int j = 0; j < wb; j++) {
        scanf("%d",&b[i][j]);
    }
  }

  int *d_a,*d_b,*d_c;
  int size = sizeof(int); 

  hipMalloc((void**)&d_a,size*ha*wa);
  hipMalloc((void**)&d_b,size*wa*wb);
  hipMalloc((void**)&d_c,size*ha*wb);

  hipMemcpy(d_a,&a,size*ha*wa,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size*wa*wb,hipMemcpyHostToDevice);

  dim3 block(wb,ha,1);
  mult<<<1,block>>>(d_a,d_b,d_c,wa,wb);
  
  hipMemcpy(&c,d_c,size*ha*wb,hipMemcpyDeviceToHost);

  printf("C:\n");
  for(int i=0; i<ha; i++) {
    for(int j=0; j<wb; j++)
      printf("%d ",c[i][j]);
    printf("\n");
  }
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
