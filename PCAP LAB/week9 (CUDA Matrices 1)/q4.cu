#include<stdio.h>
#include<hip/hip_runtime.h>


__device__ int invBin(int x) {
  int arr[100];

  int temp = x;
  int val = 0,i = 0;

  while(temp > 0) {
    arr[i++] = temp%2;
    temp = temp / 2;
  }

  for(int j=i-1; j>=0; j--)
    val = val*10 + !arr[j];

  return val;
}

__global__ void add(int *a,int *b,int m,int n) {
  int row = threadIdx.x;
  int col = threadIdx.y;
  if(row == 0 || row == m-1 || col == 0 || col == n-1) {
    b[row*n+col] = a[row*n+col];
  }
  else {
    b[row*n+col] = invBin(a[row*n+col]);
  }
}

int main() {
  int m,n,i,j;
  int *d_a,*d_b;

  printf("Enter value of m: ");
  scanf("%d",&m);
  printf("Enter value of n: ");
  scanf("%d",&n);

  int size = sizeof(int)*n*m;
  int a[m][n];
  int b[m][n];

  printf("Enter input matrix 1: ");
  for(i = 0; i<m; i++) {
    for(j = 0; j<n; j++) {
      scanf("%d",&a[i][j]);
    }   
  }

  hipMalloc((void **)&d_a,size);
  hipMalloc((void **)&d_b,size);

  hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

  dim3 block(m,n,1);
  add<<<1,block>>>(d_a,d_b,m,n);

  hipMemcpy(&b,d_b,size,hipMemcpyDeviceToHost);

  printf("Result matrix is: \n");
  for(i = 0; i < m; i++) {
    for(j = 0; j < n; j++) {
      printf("%d\t",b[i][j]);
    }
    printf("\n");
  }

  hipFree(d_a);
  hipFree(d_b);
}
