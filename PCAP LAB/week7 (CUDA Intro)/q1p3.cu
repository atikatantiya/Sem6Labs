#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b,int *c) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main() {
	printf("Enter the number of elements: (multiple of 5) ");
	int n;
	scanf("%d",&n);
	int *a = (int *)malloc(sizeof(int)*n);
	int *b = (int *)malloc(sizeof(int)*n);
	int *c = (int *)malloc(sizeof(int)*n);

	int *d_a = (int *)malloc(sizeof(int)*n);
	int *d_b = (int *)malloc(sizeof(int)*n);
	int *d_c = (int *)malloc(sizeof(int)*n);
	int size = n*sizeof(int);

	//allocate space for device copies
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//setup i/p
	for(int i=0; i<n; i++) {
		a[i] = i;
		b[i] = i+10;
	}

	//copy ips to device
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	//launch add()
	int blocks = n/5;
	add<<<blocks,5>>>(d_a,d_b,d_c);

	//copy result to host
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	for(int i=0; i<n; i++) {
		printf("%d\t",c[i]);
	}
	printf("\n");

	//cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}