#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b,int *c) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	b[idx] = (a[idx]*(*c)) + b[idx];
}

int main() {
	printf("Enter the number of elements: ");
	int n;
	scanf("%d",&n);
	int *a = (int *)malloc(sizeof(int)*n);
	int *b = (int *)malloc(sizeof(int)*n);

	int alpha;
	printf("Enter value of alpha: ");
	scanf("%d",&alpha);

	int *d_a = (int *)malloc(sizeof(int)*n);
	int *d_b = (int *)malloc(sizeof(int)*n);
	int size = n*sizeof(int);
	
	int size2 = sizeof(int);
	int *d_alpha;

	//allocate space for device copies
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_alpha,size2);


	//setup i/p
	for(int i=0; i<n; i++) {
		a[i] = i;
		b[i] = i+10;
	}

	//copy ips to device
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	hipMemcpy(d_alpha,&alpha,size2,hipMemcpyHostToDevice);

	//launch add()
	add<<<n,1>>>(d_a,d_b,d_alpha);

	//copy result to host
	hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost);
	for(int i=0; i<n; i++) {
		printf("%d\t",b[i]);
	}
	printf("\n");

	//cleanup

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_alpha);
}