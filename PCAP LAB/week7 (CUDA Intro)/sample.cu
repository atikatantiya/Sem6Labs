#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b,int *c) {
	*c = *a + *b;
}

int main() {
	int a,b,c;
	int *d_a,*d_b,*d_c;
	int size = sizeof(int);

	//allocate space for device copies
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//setup i/p
	a = 4;
	b = 5;

	//copy ips to device
	hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);

	//launch add()
	add<<<1,1>>>(d_a,d_b,d_c);

	//copy result to host
	hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
	printf("Result: %d\n",c);

	//cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}