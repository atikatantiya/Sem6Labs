#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(float *a,float *b) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;		
	b[idx] = sinf(a[idx]);
}

int main() {

	printf("Enter the number of elements: ");
	int n;
	scanf("%d",&n);

	float *a = (float *)malloc(sizeof(float)*n);
	float *b = (float *)malloc(sizeof(float)*n);

	//setup i/p
	printf("Enter radian values: ");
	for(int i=0; i<n; i++) {
		scanf("%f",&a[i]);
	}

	float *d_a = (float *)malloc(sizeof(float)*n);
	float *d_b = (float *)malloc(sizeof(float)*n);

	int size = n*sizeof(float);

	//allocate space for device copies
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);	

	//copy ips to device
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

	//launch add()
	add<<<n,1>>>(d_a,d_b);

	//copy result to host
	hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost);
	for(int i=0; i<n; i++) {
		printf("%f\t",b[i]);
	}
	printf("\n");

	//cleanup
	hipFree(d_a);
	hipFree(d_b);
}